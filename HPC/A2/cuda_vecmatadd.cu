
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
__global__ void add(int *a,int*b,int *c,int n)
{
  int row=blockIdx.x;
  int sum=0;
  for(int i=0;i<n;i++)
  {
    sum=sum+a[row*n+i]*b[i];
  }
  c[row]=sum;
}

int main()
{
  cout<<"Enter size of matrix";
  int n;
  cin>>n;
  int a[n][n],b[n],c[n];
  for(int i=0;i<n;i++)
  {
    for(int j=0;j<n;j++)
    {
    cin>>a[i][j];
  }
}
  cout<<"Enter the vector";
  for(int i=0;i<n;i++)
  {
    cin>>b[i];
  }
  int *ad,*bd,*cd;
  int size,size1;
  size=n*sizeof(int);
  size1=n*n*sizeof(int);
  hipMalloc(&ad,size1);
  hipMalloc(&bd,size);
  hipMalloc(&cd,size);
  hipMemcpy(ad,a,size1,hipMemcpyHostToDevice);
  hipMemcpy(bd,b,size,hipMemcpyHostToDevice);
  hipEvent_t start,end;
  dim3 grid(n,1);
  dim3 block(1,1);
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);
  add <<<grid,block>>>(ad,bd,cd,n);
  hipEventRecord(end);
  float time=0;
  hipEventElapsedTime(&time,start,end);
  hipMemcpy(c,cd,size,hipMemcpyDeviceToHost);
  for(int i=0;i<n;i++)
  {
  cout<<c[i]<<endl;
  }
  cout<<"The time required is"<<time<<endl;

  }
