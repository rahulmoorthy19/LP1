
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
__global__ void add(int *a,int*b,int *c,int n)
{
  int index=blockIdx.x*blockDim.x+threadIdx.x;
  if(index<n)
  {
    c[index]=a[index]+b[index];
  }
}

int main()
{
  cout<<"Enter size of vector";
  int n;
  cin>>n;
  int a[n],b[n],c[n];
  for(int i=0;i<n;i++)
  {
    cin>>a[i];
    b[i]=a[i];
  }
  int *ad,*bd,*cd;
  int size;
  size=n*sizeof(int);
  hipMalloc(&ad,size);
  hipMalloc(&bd,size);
  hipMalloc(&cd,size);
  hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
  hipMemcpy(bd,b,size,hipMemcpyHostToDevice);
  hipEvent_t start,end;
  dim3 grid(256,1);
  dim3 block(32,1);
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);
  add <<<grid,block>>>(ad,bd,cd,n);
  hipEventRecord(end);
  float time=0;
  hipEventElapsedTime(&time,start,end);
  hipMemcpy(c,cd,size,hipMemcpyDeviceToHost);
  for(int i=0;i<n;i++)
  {
  cout<<c[i]<<endl;
  }
  cout<<"The time required is"<<time<<endl;

  }
