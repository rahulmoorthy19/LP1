
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
__global__ void add(int *a,int*b,int *c,int n)
{
  int row=blockIdx.y*blockDim.y+threadIdx.y;
  int col=blockIdx.x*blockDim.x+threadIdx.x;
  int sum=0;
  for(int i=0;i<n;i++)
  {
    sum=sum+a[row*n+i]*b[i*n+col];
  }
  c[row*n+col]=sum;
}

int main()
{
  cout<<"Enter size of matrix";
  int n;
  cin>>n;
  int a[n][n],b[n][n],c[n][n];
  for(int i=0;i<n;i++)
  {
    for(int j=0;j<n;j++)
    {
    cin>>a[i][j];
  }
}
  cout<<"Enter the 2nd matrix";
  for(int i=0;i<n;i++)
  {
    for(int j=0;j<n;j++)
    {
    cin>>b[i][j];
  }
}
  int *ad,*bd,*cd;
  int size;
  size=n*n*sizeof(int);
  hipMalloc(&ad,size);
  hipMalloc(&bd,size);
  hipMalloc(&cd,size);
  hipMemcpy(ad,a,size,hipMemcpyHostToDevice);
  hipMemcpy(bd,b,size,hipMemcpyHostToDevice);
  hipEvent_t start,end;
  dim3 grid(n,n,n);
  dim3 block(1,1,1);
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);
  add <<<grid,size>>>(ad,bd,cd,n);
  hipEventRecord(end);
  float time=0;
  hipEventElapsedTime(&time,start,end);
  hipMemcpy(c,cd,size,hipMemcpyDeviceToHost);

  	for(int i=0;i<n;i++)
  	{
  		for(int j=0;j<n;j++)
  		{
  		cout<<c[i][j]<<" ";
  		}
  		cout<<endl;
  	}
  cout<<"The time required is"<<time<<endl;

  }
