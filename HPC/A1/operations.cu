
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
#define size 256
#define ssize size*4

__global__ void max_reduction(int *v,int *v_r)
{
  __shared__ int partial_sum[ssize];
  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  partial_sum[threadIdx.x]=v[tid];
  __syncthreads();
  for(int i=blockDim.x/2;i>0;i=i/2)
  {
    if(threadIdx.x<i)
    {
      partial_sum[threadIdx.x]=max(partial_sum[threadIdx.x],partial_sum[threadIdx.x+i]);
    }
    __syncthreads();
  }
  if(threadIdx.x==0)
  {
    v_r[blockIdx.x]=partial_sum[0];
  }
}

__global__ void min_reduction(int *v,int *v_r)
{
  __shared__ int partial_sum[ssize];
  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  partial_sum[threadIdx.x]=v[tid];
  __syncthreads();
  for(int i=blockDim.x/2;i>0;i=i/2)
  {
    if(threadIdx.x<i)
    {
      partial_sum[threadIdx.x]=min(partial_sum[threadIdx.x],partial_sum[threadIdx.x+i]);
    }
    __syncthreads();
  }
  if(threadIdx.x==0)
  {
    v_r[blockIdx.x]=partial_sum[0];
  }
}

__global__ void sum_reduction(int *v,int *v_r)
{
  __shared__ int partial_sum[ssize];
  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  partial_sum[threadIdx.x]=v[tid];
  __syncthreads();
  for(int i=blockDim.x/2;i>0;i=i/2)
  {
    if(threadIdx.x<i)
    {
      partial_sum[threadIdx.x]+=partial_sum[threadIdx.x+i];
    }
    __syncthreads();
  }
  if(threadIdx.x==0)
  {
    v_r[blockIdx.x]=partial_sum[0];
  }
}

__global__ void variance_reduction(int *v,int *v_r,float *mean)
{
  __shared__ int partial_sum[ssize];
  int tid=blockIdx.x*blockDim.x+threadIdx.x;
  partial_sum[threadIdx.x]=v[tid];
  __syncthreads();
  partial_sum[threadIdx.x]=(partial_sum[threadIdx.x]-*mean)*(partial_sum[threadIdx.x]-*mean);
  __syncthreads();
  for(int i=blockDim.x/2;i>0;i=i/2)
  {
    if(threadIdx.x<i)
    {
      partial_sum[threadIdx.x]+=partial_sum[threadIdx.x+i];
    }
    __syncthreads();
  }
  if(threadIdx.x==0)
  {
    v_r[blockIdx.x]=partial_sum[0];
  }
}

int main()
{
  int n = size*size;
  int blockthread=size;
  int no_block=n/blockthread;
  int *a;
  int *a_gpu;
  int *b_gpu;
  int *b;
  float time;

  hipMalloc(&a_gpu,n*sizeof(int));
  hipMalloc(&b_gpu,no_block*sizeof(int));
  a=(int*)malloc(n*sizeof(int));
  b=(int*)malloc(no_block*sizeof(int));
  for(int i =0;i<n;i++){
    a[i]= rand()%1000;
  }
  hipMemcpy(a_gpu,a,n*sizeof(int),hipMemcpyHostToDevice);
  hipEvent_t start,stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  max_reduction<<<no_block,blockthread>>>(a_gpu,b_gpu);
  max_reduction<<<1,blockthread>>>(b_gpu,b_gpu);
  hipMemcpy(b,b_gpu,blockthread*sizeof(int),hipMemcpyDeviceToHost);
  hipEventRecord(stop);
  hipEventElapsedTime(&time,start,stop);
  cout<<b[0]<<"\n";
  cout<<time;
}
